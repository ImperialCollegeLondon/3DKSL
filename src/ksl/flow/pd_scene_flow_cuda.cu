#include "hip/hip_runtime.h"
/*****************************************************************************
**				Primal-Dual Scene Flow for RGB-D cameras					**
**				----------------------------------------					**
**																			**
**	Copyright(c) 2015, Mariano Jaimez Tarifa, University of Malaga			**
**	Copyright(c) 2015, Mohamed Souiai, Technical University of Munich		**
**	Copyright(c) 2015, MAPIR group, University of Malaga					**
**	Copyright(c) 2015, Computer Vision group, Tech. University of Munich	**
**																			**
**  This program is free software: you can redistribute it and/or modify	**
**  it under the terms of the GNU General Public License (version 3) as		**
**	published by the Free Software Foundation.								**
**																			**
**  This program is distributed in the hope that it will be useful, but		**
**	WITHOUT ANY WARRANTY; without even the implied warranty of				**
**  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the			**
**  GNU General Public License for more details.							**
**																			**
**  You should have received a copy of the GNU General Public License		**
**  along with this program.  If not, see <http://www.gnu.org/licenses/>.	**
**																			**
*****************************************************************************/

#include <ksl/flow/pd_scene_flow_cuda.h>

//                         Memory allocation - device
//=============================================================================
__host__ void CSF_cuda::allocateDevMemory()
{
    const unsigned int width = 640/cam_mode;
    const unsigned int height = 480/cam_mode;
    unsigned int s;

    //Allocate the unfiltered depth and colour images on GPU
    hipError_t err = hipMalloc((void**)&depth_wf_dev, width*height*sizeof(float) );
    //printf("%s", hipGetErrorString(err));
    hipMalloc((void**)&colour_wf_dev, width*height*sizeof(float) );

    //Resize pyramid. Allocate memory for the different levels
    const unsigned int pyr_levels = roundf(log2f(width/cols)) + ctf_levels;

    for (unsigned int i = 0; i<pyr_levels; i++)
    {
        s = static_cast<unsigned int>(powf(2,i));
        hipMalloc((void**)&colour_dev[i], width*height*sizeof(float)/(s*s) );
        hipMalloc((void**)&colour_old_dev[i], width*height*sizeof(float)/(s*s) );
        hipMalloc((void**)&depth_dev[i], width*height*sizeof(float)/(s*s) );
        hipMalloc((void**)&depth_old_dev[i], width*height*sizeof(float)/(s*s) );
        hipMalloc((void**)&xx_dev[i], width*height*sizeof(float)/(s*s) );
        hipMalloc((void**)&xx_old_dev[i], width*height*sizeof(float)/(s*s) );
        hipMalloc((void**)&yy_dev[i], width*height*sizeof(float)/(s*s) );
        hipMalloc((void**)&yy_old_dev[i], width*height*sizeof(float)/(s*s) );
    }

    //Allocate dx, dy, dz on GPU
    hipMalloc((void**)&dx_dev, sizeof(float) );
    hipMalloc((void**)&dy_dev, sizeof(float) );
    hipMalloc((void**)&dz_dev, sizeof(float) );

    //Allocate final solutions at the biggest resolution only once
    hipMalloc((void**)&du_l_dev, width*height*sizeof(float) );
    hipMalloc((void**)&dv_l_dev, width*height*sizeof(float) );
    hipMalloc((void**)&dw_l_dev, width*height*sizeof(float) );
    hipMalloc((void**)&pd_l_dev, width*height*sizeof(float) );
    hipMalloc((void**)&puu_l_dev, width*height*sizeof(float) );
    hipMalloc((void**)&puv_l_dev, width*height*sizeof(float) );
    hipMalloc((void**)&pvu_l_dev, width*height*sizeof(float) );
    hipMalloc((void**)&pvv_l_dev, width*height*sizeof(float) );
    hipMalloc((void**)&pwu_l_dev, width*height*sizeof(float) );
    hipMalloc((void**)&pwv_l_dev, width*height*sizeof(float) );
}

__host__ void CSF_cuda::allocateMemoryNewLevel(unsigned int rows_loc, unsigned int cols_loc, unsigned int level_i, unsigned int level_image_i)
{
    local_level = level_i;
    level_image = level_image_i;
    rows_i = rows_loc;
    cols_i = cols_loc;

    //Allocate derivatives on GPU
    hipMalloc((void**)&dct_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&dcu_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&dcv_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&ddt_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&ddu_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&ddv_dev, rows_i*cols_i*sizeof(float) );

    hipMalloc((void**)&dcu_aux_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&dcv_aux_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&ddu_aux_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&ddv_aux_dev, rows_i*cols_i*sizeof(float) );

    //Allocate gradients on GPU
    hipMalloc((void**)&gradu1_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&gradu2_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&gradv1_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&gradv2_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&gradw1_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&gradw2_dev, rows_i*cols_i*sizeof(float) );

    //Allocate divergence on GPU
    hipMalloc((void**)&divpu_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&divpv_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&divpw_dev, rows_i*cols_i*sizeof(float) );

    //Allocate step sizes on GPU
    hipMalloc((void**)&sigma_pd_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&sigma_puvx_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&sigma_puvy_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&sigma_pwx_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&sigma_pwy_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&tau_u_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&tau_v_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&tau_w_dev, rows_i*cols_i*sizeof(float) );

    //Allocate mu_uv on GPU
    hipMalloc((void**)&mu_uv_dev, rows_i*cols_i*sizeof(float) );

    //Allocate du_acc, dv_acc, dw_acc on GPU
    hipMalloc((void**)&du_acc_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&dv_acc_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&dw_acc_dev, rows_i*cols_i*sizeof(float) );

    //Allocate ri, rj, ri_2, rj_2, du_prev, dv_prev on GPU
    hipMalloc((void**)&ri_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&rj_dev, rows_i*cols_i*sizeof(float) );
	hipMalloc((void**)&ri_2_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&rj_2_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&du_prev_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&dv_prev_dev, rows_i*cols_i*sizeof(float) );

    //Allocate values of previous level on GPU
    hipMalloc((void**)&du_new_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&dv_new_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&dw_new_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&pd_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&puu_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&puv_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&pvu_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&pvv_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&pwu_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&pwv_dev, rows_i*cols_i*sizeof(float) );

    //Allocate memory for the upsampling variables
    hipMalloc((void**)&du_upsamp_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&dv_upsamp_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&dw_upsamp_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&pd_upsamp_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&puu_upsamp_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&puv_upsamp_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&pvu_upsamp_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&pvv_upsamp_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&pwu_upsamp_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&pwv_upsamp_dev, rows_i*cols_i*sizeof(float) );

    //Allocate dx, dy, dz on GPU
    hipFree(dx_dev); hipFree(dy_dev); hipFree(dz_dev);
    hipMalloc((void**)&dx_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&dy_dev, rows_i*cols_i*sizeof(float) );
    hipMalloc((void**)&dz_dev, rows_i*cols_i*sizeof(float) );
}

//                          Copy object to device
//=============================================================================
CSF_cuda *ObjectToDevice(CSF_cuda *csf_host)
{
    CSF_cuda *csf_device;
    hipMalloc((void**)&csf_device, sizeof(CSF_cuda) );
    hipMemcpy(csf_device, csf_host, sizeof(CSF_cuda), hipMemcpyHostToDevice);
    return csf_device;
}

//                Copy data from host to device and viceversa
//=============================================================================
__host__ void CSF_cuda::readParameters(unsigned int rows_host, unsigned int cols_host, float lambda_i_host, float lambda_d_host, float mu_host,
									   float *g_mask, unsigned int levels_host, unsigned int cam_mode_host, float fovh_host, float fovv_host)
{
    rows = rows_host;
    cols = cols_host;
    lambda_i = lambda_i_host;
    lambda_d = lambda_d_host;
    mu = mu_host;
    ctf_levels = levels_host;
    cam_mode = cam_mode_host;
    fovh = fovh_host;
    fovv = fovv_host;

    //Allocate  and copy gaussian mask
    hipError_t err = hipMalloc((void**)&g_mask_dev, 5*5*sizeof(float));
    //printf("%s", hipGetErrorString(err));
    hipMemcpy(g_mask_dev, g_mask, 5*5*sizeof(float), hipMemcpyHostToDevice);
}

__host__ void CSF_cuda::copyNewFrames(float *colour_wf, float *depth_wf)
{
    const unsigned int width = 640/cam_mode;
    const unsigned int height = 480/cam_mode;

    hipMemcpy(depth_wf_dev, depth_wf, width*height*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(colour_wf_dev, colour_wf, width*height*sizeof(float), hipMemcpyHostToDevice);

	//Swap pointers of old and new images of the pyramid (equivalent to pushing the new frames to the old ones)
	for (unsigned int i=0; i<8; i++)
	{
		float *temp = colour_old_dev[i];
		colour_old_dev[i] = colour_dev[i];
		colour_dev[i] = temp;

		temp = depth_old_dev[i];
		depth_old_dev[i] = depth_dev[i];
		depth_dev[i] = temp;

		temp = xx_old_dev[i];
		xx_old_dev[i] = xx_dev[i];
		xx_dev[i] = temp;

		temp = yy_old_dev[i];
		yy_old_dev[i] = yy_dev[i];
		yy_dev[i] = temp;
	}	
}

__host__ void CSF_cuda::copyAllSolutions(float *dx, float *dy, float *dz, float *depth, float *depth_old, float *colour, float *colour_old, float *xx, float *xx_old, float *yy, float *yy_old)
{
    hipMemcpy(dx, dx_dev, rows_i*cols_i*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(dy, dy_dev, rows_i*cols_i*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(dz, dz_dev, rows_i*cols_i*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(depth, depth_dev[level_image], rows_i*cols_i*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(depth_old, depth_old_dev[level_image], rows_i*cols_i*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(colour, colour_dev[level_image], rows_i*cols_i*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(colour_old, colour_old_dev[level_image], rows_i*cols_i*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(xx, xx_dev[level_image], rows_i*cols_i*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(xx_old, xx_old_dev[level_image], rows_i*cols_i*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(yy, yy_dev[level_image], rows_i*cols_i*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(yy_old, yy_old_dev[level_image], rows_i*cols_i*sizeof(float), hipMemcpyDeviceToHost);
}

__host__ void CSF_cuda::copyMotionField(float *dx, float *dy, float *dz)
{
    hipMemcpy(dx, dx_dev, rows_i*cols_i*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(dy, dy_dev, rows_i*cols_i*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(dz, dz_dev, rows_i*cols_i*sizeof(float), hipMemcpyDeviceToHost);
}


//                              Free memory - device
//=============================================================================
__host__ void CSF_cuda::freeDeviceMemory()
{
    hipFree(g_mask_dev);

    const unsigned int width = 640/cam_mode;
    const unsigned int pyr_levels = roundf(log2f(width/cols)) + ctf_levels;
    for (unsigned int i = 0; i<pyr_levels; i++)
    {
        hipFree(depth_old_dev[i]);
        hipFree(depth_dev[i]);
        hipFree(colour_old_dev[i]);
        hipFree(colour_dev[i]);
        hipFree(xx_old_dev[i]);
        hipFree(xx_dev[i]);
        hipFree(yy_old_dev[i]);
        hipFree(yy_dev[i]);
    }

    //Free pointers to pointers
    hipFree(depth_old_dev);
    hipFree(depth_dev);
    hipFree(colour_old_dev);
    hipFree(colour_dev);
    hipFree(xx_old_dev);
    hipFree(xx_dev);
    hipFree(yy_old_dev);
    hipFree(yy_dev);

    hipFree(du_l_dev); hipFree(dv_l_dev); hipFree(dw_l_dev);
    hipFree(pd_l_dev);
    hipFree(puu_l_dev); hipFree(puv_l_dev);
    hipFree(pvu_l_dev); hipFree(pvv_l_dev);
    hipFree(pwu_l_dev); hipFree(pwv_l_dev);

    hipFree(dx_dev); hipFree(dy_dev); hipFree(dz_dev);
}

__host__ void CSF_cuda::freeLevelVariables()
{
    hipFree(du_upsamp_dev); hipFree(dv_upsamp_dev); hipFree(dw_upsamp_dev);
    hipFree(pd_upsamp_dev);
    hipFree(puu_upsamp_dev); hipFree(puv_upsamp_dev);
    hipFree(pvu_upsamp_dev); hipFree(pvv_upsamp_dev);
    hipFree(pwu_upsamp_dev); hipFree(pwv_upsamp_dev);

    hipFree(du_new_dev);
    hipFree(dv_new_dev);
    hipFree(dw_new_dev);
    hipFree(pd_dev);
    hipFree(puu_dev); hipFree(puv_dev);
    hipFree(pvu_dev); hipFree(pvv_dev);
    hipFree(pwu_dev); hipFree(pwv_dev);

    hipFree(mu_uv_dev);
    hipFree(ri_dev); hipFree(rj_dev);
    hipFree(ri_2_dev); hipFree(rj_2_dev);
    hipFree(du_acc_dev); hipFree(dv_acc_dev); hipFree(dw_acc_dev);
    hipFree(du_prev_dev); hipFree(dv_prev_dev);

    hipFree(dct_dev); hipFree(dcu_dev); hipFree(dcv_dev);
    hipFree(ddt_dev); hipFree(ddu_dev); hipFree(ddv_dev);
    hipFree(dcu_aux_dev); hipFree(dcv_aux_dev);
    hipFree(ddu_aux_dev); hipFree(ddv_aux_dev);

    hipFree(gradu1_dev); hipFree(gradu2_dev);
    hipFree(gradv1_dev); hipFree(gradv2_dev);
    hipFree(gradw1_dev); hipFree(gradw2_dev);

    hipFree(divpu_dev); hipFree(divpv_dev); hipFree(divpw_dev);

    hipFree(sigma_pd_dev); hipFree(sigma_puvx_dev); hipFree(sigma_puvy_dev);
    hipFree(sigma_pwx_dev); hipFree(sigma_pwy_dev);
    hipFree(tau_u_dev); hipFree(tau_v_dev); hipFree(tau_w_dev);
}


//                  Create gaussian pyramid
//=============================================================================
__device__ void CSF_cuda::computePyramidLevel(unsigned int index, unsigned int level)
{
    //Shared memory for the gaussian mask
	__shared__ float mask_shared[25];
	if (threadIdx.x < 25)			//Warning!!!!!! Number of threads should be higher than 25
		mask_shared[threadIdx.x] = g_mask_dev[threadIdx.x];
	__syncthreads();

	
	const float max_depth_dif = 0.1f;

    //Calculate indices
    const unsigned int v = index%(rows_i);
    const unsigned int u = index/(rows_i);

    if (level == 0)
    {
        //Copy intensity image
        colour_dev[level][index] = colour_wf_dev[index];

        //Copy depth image
        depth_dev[level][index] = depth_wf_dev[index];
    }

    //                              Downsampling
    //-----------------------------------------------------------------------------
    else
    {
        float sumd = 0.f, sumc = 0.f, acu_weights_d = 0.f, acu_weights_c = 0.f;
		const unsigned int ind_cent_prev = 2*v + 4*u*rows_i;
		const float dcenter = depth_dev[level-1][ind_cent_prev];
		
		//Inner pixels
        if ((v>0)&&(v<rows_i-1)&&(u>0)&&(u<cols_i-1))
        {	
			for (int k=-2; k<3; k++)
                for (int l=-2; l<3; l++)
                {
                    const unsigned int ind_loop_prev = 2*v+k + 2*(2*u+l)*rows_i;
                    const unsigned int ind_mask = 12+k+5*l;	//2+k+(2+l)*5

                    //Colour
                    sumc += mask_shared[ind_mask]*colour_dev[level-1][ind_loop_prev];

                    //Depth
                    if ((depth_dev[level-1][ind_loop_prev] > 0.f)&&(fabsf(depth_dev[level-1][ind_loop_prev]-dcenter) < max_depth_dif))
                    {
                        const float aux_w = mask_shared[ind_mask]*(max_depth_dif - fabsf(depth_dev[level-1][ind_loop_prev] - dcenter));
                        acu_weights_d += aux_w;
                        sumd += aux_w*depth_dev[level-1][ind_loop_prev];
                    }
                }

            if (sumd > 0.f)
                depth_dev[level][index] = sumd/acu_weights_d;
			else
				depth_dev[level][index] = 0.f;

			colour_dev[level][index] = sumc;
        }

        //Boundary
        else
        {
            for (int k=-2; k<3; k++)
                for (int l=-2; l<3; l++)
                {
                    const int indv = 2*v+k, indu = 2*u+l;
                    if ((indv>=0)&&(indv<2*rows_i)&&(indu>=0)&&(indu<2*cols_i))
                    {
                        const unsigned int ind_loop_prev = 2*v+k + 2*(2*u+l)*rows_i;
                        const unsigned int ind_mask = 12+k+5*l;	//2+k+(2+l)*5

                        //Colour
                        sumc += mask_shared[ind_mask]*colour_dev[level-1][ind_loop_prev];
                        acu_weights_c += mask_shared[ind_mask];

                        //Depth
                        if ((depth_dev[level-1][ind_loop_prev] > 0.f)&&(fabsf(depth_dev[level-1][ind_loop_prev]-depth_dev[level-1][ind_cent_prev]) < max_depth_dif))
                        {
                            const float aux_w = mask_shared[ind_mask]*(max_depth_dif - fabsf(depth_dev[level-1][ind_loop_prev] - depth_dev[level-1][ind_cent_prev]));
                            acu_weights_d += aux_w;
                            sumd += aux_w*depth_dev[level-1][ind_loop_prev];
                        }
                    }
                }

            colour_dev[level][index] = sumc/acu_weights_c;

            if (sumd > 0.f)
                depth_dev[level][index] = sumd/acu_weights_d;
			else
				depth_dev[level][index] = 0.f;
        }
    }

    //Calculate coordinates "xy" of the points
    const float inv_f_i = 2.f*tan(0.5f*fovh)/float(cols_i);
    const float disp_u_i = 0.5f*(cols_i-1);
    const float disp_v_i = 0.5f*(rows_i-1);

    xx_dev[level][index] = (u - disp_u_i)*depth_dev[level][index]*inv_f_i;
    yy_dev[level][index] = (v - disp_v_i)*depth_dev[level][index]*inv_f_i;
}


//                  Initiallize some variables
//=============================================================================
__device__ void CSF_cuda::assignZeros(unsigned int index)
{
    du_upsamp_dev[index] = 0.f; dv_upsamp_dev[index] = 0.f; dw_upsamp_dev[index] = 0.f;
    pd_upsamp_dev[index] = 0.f;
    puu_upsamp_dev[index] = 0.f; puv_upsamp_dev[index] = 0.f;
    pvu_upsamp_dev[index] = 0.f; pvv_upsamp_dev[index] = 0.f;
    pwu_upsamp_dev[index] = 0.f; pwv_upsamp_dev[index] = 0.f;

    du_prev_dev[index] = 0.f; dv_prev_dev[index] = 0.f;
    du_new_dev[index] = 0.f; dv_new_dev[index] = 0.f; dw_new_dev[index] = 0.f;
    pd_dev[index] = 0.f;
    puu_dev[index] = 0.f; puv_dev[index] = 0.f;
    pvu_dev[index] = 0.f; pvv_dev[index] = 0.f;
    pwu_dev[index] = 0.f; pwv_dev[index] = 0.f;

    du_acc_dev[index] = 0.f; dv_acc_dev[index] = 0.f; dw_acc_dev[index] = 0.f;
}


//                  Upsample previous solution
//=============================================================================
__device__ void CSF_cuda::upsampleCopyPrevSolution(unsigned int index)
{
    //Calculate (v,u)
    const unsigned int v = index%(rows_i/2);
    const unsigned int u = 2*index/(rows_i);
    const unsigned int index_big = 2*(v + u*rows_i);

    du_upsamp_dev[index_big] = 2.f*du_l_dev[index];
    dv_upsamp_dev[index_big] = 2.f*dv_l_dev[index];
    dw_upsamp_dev[index_big] = dw_l_dev[index];
    pd_upsamp_dev[index_big] = pd_l_dev[index];
    puu_upsamp_dev[index_big] = puu_l_dev[index];
    puv_upsamp_dev[index_big] = puv_l_dev[index];
    pvu_upsamp_dev[index_big] = pvu_l_dev[index];
    pvv_upsamp_dev[index_big] = pvv_l_dev[index];
    pwu_upsamp_dev[index_big] = pwu_l_dev[index];
    pwv_upsamp_dev[index_big] = pwv_l_dev[index];
}

__device__ void CSF_cuda::upsampleFilterPrevSolution(unsigned int index)
{
    const unsigned int v = index%rows_i;
    const unsigned int u = index/rows_i;

	//Shared memory for the gaussian mask - Warning!! The number of threads should be higher than 25
	__shared__ float mask_shared[25];
	if (threadIdx.x < 25)			
		mask_shared[threadIdx.x] = 4.f*g_mask_dev[threadIdx.x];
	__syncthreads();

	float du = 0.f, dv = 0.f, dw = 0.f, pd = 0.f, puu = 0.f, puv = 0.f, pvu = 0.f, pvv = 0.f, pwu = 0.f, pwv = 0.f;

    //Inner pixels
    if ((v>1)&&(v<rows_i-2)&&(u>1)&&(u<cols_i-2))
    {
        for (int k=-2; k<3; k++)
            for (int l=-2; l<3; l++)
            {
                const unsigned incr_index = v+k+(u+l)*rows_i;
                const float gmask_factor = mask_shared[12 + k + 5*l];	//[2+k + (2+l)*5]
                du += gmask_factor*du_upsamp_dev[incr_index];
                dv += gmask_factor*dv_upsamp_dev[incr_index];
                dw += gmask_factor*dw_upsamp_dev[incr_index];
                pd  += gmask_factor*pd_upsamp_dev[incr_index];
                puu += gmask_factor*puu_upsamp_dev[incr_index];
                puv += gmask_factor*puv_upsamp_dev[incr_index];
                pvu += gmask_factor*pvu_upsamp_dev[incr_index];
                pvv += gmask_factor*pvv_upsamp_dev[incr_index];
                pwu += gmask_factor*pwu_upsamp_dev[incr_index];
                pwv += gmask_factor*pwv_upsamp_dev[incr_index];
            }
    }
    //Boundary
    else
    {
        float acu_weight = 1.f;
        for (int k=-2; k<3; k++)
            for (int l=-2; l<3; l++)
            {
                const int indv = v+k, indu = u+l;
                if ((indv<0)||(indv>=rows_i)||(indu<0)||(indu>=cols_i))
                {
                    acu_weight -= 0.25f*mask_shared[12 + k + 5*l];		//[2+k + (2+l)*5]
                    continue;
                }
                else
                {
                    const unsigned incr_index = v+k+(u+l)*rows_i;
                    const float gmask_factor = mask_shared[12 + k + 5*l];	//[2+k + (2+l)*5]
                    du += gmask_factor*du_upsamp_dev[incr_index];
                    dv += gmask_factor*dv_upsamp_dev[incr_index];
                    dw += gmask_factor*dw_upsamp_dev[incr_index];
                    pd  += gmask_factor*pd_upsamp_dev[incr_index];
                    puu += gmask_factor*puu_upsamp_dev[incr_index];
                    puv += gmask_factor*puv_upsamp_dev[incr_index];
                    pvu += gmask_factor*pvu_upsamp_dev[incr_index];
                    pvv += gmask_factor*pvv_upsamp_dev[incr_index];
                    pwu += gmask_factor*pwu_upsamp_dev[incr_index];
                    pwv += gmask_factor*pwv_upsamp_dev[incr_index];
                }
            }

		const float inv_acu_weight = fdividef(1.f, acu_weight);
        du *= inv_acu_weight;
        dv *= inv_acu_weight;
        dw *= inv_acu_weight;
        pd  *= inv_acu_weight;
        puu *= inv_acu_weight;
        puv *= inv_acu_weight;
        pvu *= inv_acu_weight;
        pvv *= inv_acu_weight;
        pwu *= inv_acu_weight;
        pwv *= inv_acu_weight;
    }

	//Write results to global memory
	du_prev_dev[index] = du;
    dv_prev_dev[index] = dv;
    dw_new_dev[index]  = dw;
    pd_dev[index]  = pd;
    puu_dev[index] = puu;
    puv_dev[index] = puv;
    pvu_dev[index] = pvu;
    pvv_dev[index] = pvv;
    pwu_dev[index] = pwu;
    pwv_dev[index] = pwv;

    //Last update, for dw_acc
    dw_acc_dev[index] = dw;
}


//                  Compute intensity and depth derivatives
//=============================================================================
__device__ void CSF_cuda::computeImGradients(unsigned int index)
{
    //Calculate (v,u)
    const unsigned int v = index%rows_i;
    const unsigned int u = index/rows_i;

    //Row gradients
    if (u == 0)
    {
        dcu_aux_dev[index] = colour_dev[level_image][index+rows_i] - colour_dev[level_image][index];
        ddu_aux_dev[index] = depth_dev[level_image][index+rows_i] - depth_dev[level_image][index];
    }
    else if (u == cols_i-1)
    {
        dcu_aux_dev[index] = colour_dev[level_image][index] - colour_dev[level_image][index-rows_i];
        ddu_aux_dev[index] = depth_dev[level_image][index] - depth_dev[level_image][index-rows_i];
    }
    else
    {
		dcu_aux_dev[index] = (ri_2_dev[index]*(colour_dev[level_image][index+rows_i]-colour_dev[level_image][index])
							+ ri_2_dev[index-rows_i]*(colour_dev[level_image][index]-colour_dev[level_image][index-rows_i]))
							/(ri_2_dev[index]+ri_2_dev[index-rows_i]);
		if (depth_dev[level_image][index] > 0.f)
			ddu_aux_dev[index] = (ri_2_dev[index]*(depth_dev[level_image][index+rows_i]-depth_dev[level_image][index])
								+ ri_2_dev[index-rows_i]*(depth_dev[level_image][index]-depth_dev[level_image][index-rows_i]))
								/(ri_2_dev[index]+ri_2_dev[index-rows_i]);
		else
			ddu_aux_dev[index] = 0.f;
    }

    //Col gradients
    if (v == 0)
    {
        dcv_aux_dev[index] = colour_dev[level_image][index+1] - colour_dev[level_image][index];
        ddv_aux_dev[index] = depth_dev[level_image][index+1] - depth_dev[level_image][index];
    }
    else if (v == rows_i-1)
    {
        dcv_aux_dev[index] = colour_dev[level_image][index] - colour_dev[level_image][index-1];
        ddv_aux_dev[index] = depth_dev[level_image][index] - depth_dev[level_image][index-1];
    }
    else
    {
		dcv_aux_dev[index] = (rj_2_dev[index]*(colour_dev[level_image][index+1]-colour_dev[level_image][index])
							+ rj_2_dev[index-1]*(colour_dev[level_image][index]-colour_dev[level_image][index-1]))
							/(rj_2_dev[index]+rj_2_dev[index-1]);
		if (depth_dev[level_image][index] > 0.f)
			ddv_aux_dev[index] = (rj_2_dev[index]*(depth_dev[level_image][index+1]-depth_dev[level_image][index])
								+ rj_2_dev[index-1]*(depth_dev[level_image][index]-depth_dev[level_image][index-1]))
								/(rj_2_dev[index]+rj_2_dev[index-1]);
		else
			ddv_aux_dev[index] = 0.f;
    }
}

__device__ void CSF_cuda::performWarping(unsigned int index)
{
    //Calculate (v,u)
    const unsigned int v = index%rows_i;
    const unsigned int u = index/rows_i;
    float warped_pixel;

    //Intensity images
	const float ind_uf = float(u) + du_prev_dev[index];
	const float ind_vf = float(v) + dv_prev_dev[index];
    warped_pixel = interpolatePixel(colour_dev[level_image], ind_uf, ind_vf);
    dct_dev[index] = warped_pixel - colour_old_dev[level_image][index];
    dcu_dev[index] = interpolatePixel(dcu_aux_dev, ind_uf, ind_vf);
    dcv_dev[index] = interpolatePixel(dcv_aux_dev, ind_uf, ind_vf);

	//Depth images
	warped_pixel = interpolatePixelDepth(depth_dev[level_image], ind_uf, ind_vf);
	if (warped_pixel > 0.f)
		ddt_dev[index] = warped_pixel - depth_old_dev[level_image][index];
	else
		ddt_dev[index] = 0.f;
	ddu_dev[index] = interpolatePixel(ddu_aux_dev, ind_uf, ind_vf);
	ddv_dev[index] = interpolatePixel(ddv_aux_dev, ind_uf, ind_vf);
}

__device__ float CSF_cuda::interpolatePixel(float *mat, float ind_u, float ind_v)
{
    if (ind_u < 0.f) { ind_u = 0.f;}
	else if (ind_u > cols_i - 1.f) { ind_u = cols_i - 1.f;}
	if (ind_v < 0.f) { ind_v = 0.f;}
	else if (ind_v > rows_i - 1.f) { ind_v = rows_i - 1.f;}

    const unsigned int sup_u = __float2int_ru(ind_u);
    const unsigned int inf_u = __float2int_rd(ind_u);
    const unsigned int sup_v = __float2int_ru(ind_v);
    const unsigned int inf_v = __float2int_rd(ind_v);

    if ((sup_u == inf_u)&&(sup_v == inf_v))
        return mat[lrintf(ind_v + rows_i*ind_u)];

    else if (sup_u == inf_u)
        return (sup_v - ind_v)*mat[inf_v + rows_i*lrintf(ind_u)] + (ind_v - inf_v)*mat[sup_v + rows_i*lrintf(ind_u)];

    else if (sup_v == inf_v)
        return (sup_u - ind_u)*mat[lrintf(ind_v) + rows_i*inf_u] + (ind_u - inf_u)*mat[lrintf(ind_v) + rows_i*sup_u];

    else
    {
        //First in u
        const float val_sup_v = (sup_u - ind_u)*mat[sup_v + rows_i*inf_u] + (ind_u - inf_u)*mat[sup_v + rows_i*sup_u];
        const float val_inf_v = (sup_u - ind_u)*mat[inf_v + rows_i*inf_u] + (ind_u - inf_u)*mat[inf_v + rows_i*sup_u];
        return (sup_v - ind_v)*val_inf_v + (ind_v - inf_v)*val_sup_v;
    }
}

__device__ float CSF_cuda::interpolatePixelDepth(float *mat, float ind_u, float ind_v)
{
    if (ind_u < 0.f) { ind_u = 0.f;}
	else if (ind_u > cols_i - 1.f) { ind_u = cols_i - 1.f;}
	if (ind_v < 0.f) { ind_v = 0.f;}
	else if (ind_v > rows_i - 1.f) { ind_v = rows_i - 1.f;}

    const unsigned int sup_u = __float2int_ru(ind_u);
    const unsigned int inf_u = __float2int_rd(ind_u);
    const unsigned int sup_v = __float2int_ru(ind_v);
    const unsigned int inf_v = __float2int_rd(ind_v);

    if ((mat[sup_v + rows_i*sup_u] == 0.f)||(mat[sup_v + rows_i*inf_u] == 0.f)||(mat[inf_v + rows_i*sup_u] == 0.f)||(mat[inf_v + rows_i*inf_u]==0.f))
    {
        const unsigned int rind_u = __float2int_rn(ind_u);
        const unsigned int rind_v = __float2int_rn(ind_v);
        return mat[rind_v + rows_i*rind_u];
    }
    else
    {	
		if ((sup_u == inf_u)&&(sup_v == inf_v))
			return mat[lrintf(ind_v + rows_i*ind_u)];

		else if (sup_u == inf_u)
			return (sup_v - ind_v)*mat[inf_v + rows_i*lroundf(ind_u)] + (ind_v - inf_v)*mat[sup_v + rows_i*lroundf(ind_u)];

		else if (sup_v == inf_v)
			return (sup_u - ind_u)*mat[lroundf(ind_v) + rows_i*inf_u] + (ind_u - inf_u)*mat[lroundf(ind_v) + rows_i*sup_u];

		else
		{
			//First in u
			const float val_sup_v = (sup_u - ind_u)*mat[sup_v + rows_i*inf_u] + (ind_u - inf_u)*mat[sup_v + rows_i*sup_u];
			const float val_inf_v = (sup_u - ind_u)*mat[inf_v + rows_i*inf_u] + (ind_u - inf_u)*mat[inf_v + rows_i*sup_u];
			return (sup_v - ind_v)*val_inf_v + (ind_v - inf_v)*val_sup_v;
		}
	}
}

//                          Preliminary computations
//=============================================================================
__device__ void CSF_cuda::computeRij(unsigned int index)
{
    //Calculate (v,u)
    const unsigned int v = index%rows_i;
    const unsigned int u = index/rows_i;

    float dxu, dzu, dxu_2, dzu_2;
    float dyv, dzv, dyv_2, dzv_2;

    if (u == cols_i-1)
    {
        dxu = 0.f; dzu = 0.f;
		dxu_2 = 0.f; dzu_2 = 0.f;
    }
    else
    {
        dxu = xx_old_dev[level_image][index + rows_i] - xx_old_dev[level_image][index];
        dzu = depth_old_dev[level_image][index + rows_i] - depth_old_dev[level_image][index];
		dxu_2 = xx_dev[level_image][index + rows_i] - xx_dev[level_image][index];
        dzu_2 = depth_dev[level_image][index + rows_i] - depth_dev[level_image][index];
    }

    if (v == rows_i-1)
    {
        dyv = 0.f; dzv = 0.f;
		dyv_2 = 0.f; dzv_2 = 0.f;
    }
    else
    {
        dyv = yy_old_dev[level_image][index+1] - yy_old_dev[level_image][index];
        dzv = depth_old_dev[level_image][index+1] - depth_old_dev[level_image][index];
		dyv_2 = yy_dev[level_image][index+1] - yy_dev[level_image][index];
        dzv_2 = depth_dev[level_image][index+1] - depth_dev[level_image][index];
    }

    if (fabsf(dxu) + fabsf(dzu) > 0.f)
        ri_dev[index] = 2.f*rhypotf(dxu,dzu);	//2.f/sqrtf(dxu*dxu + dzu*dzu);
    else
        ri_dev[index] = 1.f;

    if (fabsf(dyv) + fabsf(dzv) > 0.f)
        rj_dev[index] = 2.f*rhypotf(dyv,dzv);	//2.f/sqrtf(dyv*dyv + dzv*dzv);
    else
        rj_dev[index] = 1.f;

	if (fabsf(dxu_2) + fabsf(dzu_2) > 0.f)
        ri_2_dev[index] = 2.f*rhypotf(dxu_2,dzu_2);	//2.f/sqrtf(dxu*dxu + dzu*dzu);
    else
        ri_2_dev[index] = 1.f;

    if (fabsf(dyv_2) + fabsf(dzv_2) > 0.f)
        rj_2_dev[index] = 2.f*rhypotf(dyv_2,dzv_2);	//2.f/sqrtf(dyv*dyv + dzv*dzv);
    else
        rj_2_dev[index] = 1.f;
}

__device__ void CSF_cuda::computeMu(unsigned int index)
{
    mu_uv_dev[index] = fdividef(mu, 1.f + 1000.f*(ddu_dev[index]*ddu_dev[index] + ddv_dev[index]*ddv_dev[index] + ddt_dev[index]*ddt_dev[index]));
}

__device__ void CSF_cuda::computeStepSizes(unsigned int index)
{
    //Load lambda from global memory
    const float lambdai = lambda_i, lambdad = lambda_d;
	
	sigma_pd_dev[index] = fdividef(1.f, mu_uv_dev[index]*(1.f + abs(ddu_dev[index]) + abs(ddv_dev[index])) + 1e-10f);
    sigma_puvx_dev[index] = fdividef(0.5f, lambdai*ri_dev[index] + 1e-10f);
    sigma_puvy_dev[index] = fdividef(0.5f, lambdai*rj_dev[index] + 1e-10f);
    sigma_pwx_dev[index] = fdividef(0.5f, ri_dev[index]*lambdad + 1e-10f);
    sigma_pwy_dev[index] = fdividef(0.5f, rj_dev[index]*lambdad + 1e-10f);

	//Calculate (v,u)
    const unsigned int v = index%rows_i;
    const unsigned int u = index/rows_i;

	float acu_r = ri_dev[index] + rj_dev[index];
	if (u > 0) acu_r += ri_dev[index-rows_i];
	if (v > 0) acu_r += rj_dev[index-1];

    tau_u_dev[index] = fdividef(1.f, mu_uv_dev[index]*abs(ddu_dev[index]) + lambdai*acu_r + 1e-10f);
    tau_v_dev[index] = fdividef(1.f, mu_uv_dev[index]*abs(ddv_dev[index]) + lambdai*acu_r + 1e-10f);
    tau_w_dev[index] = fdividef(1.f, mu_uv_dev[index] + lambdad*acu_r + 1e-10f);
}


//                              Main iteration
//=============================================================================
__device__ void CSF_cuda::updateDualVariables(unsigned int index)
{
	//Create aux variables to avoid repetitive global memory access
    float module_p;
	float pd = pd_dev[index], puu = puu_dev[index], puv = puv_dev[index];
	float pvu = pvu_dev[index], pvv = pvv_dev[index], pwu = pwu_dev[index], pwv = pwv_dev[index];

    //Update dual variables
    //Solve pd
    pd += sigma_pd_dev[index]*mu_uv_dev[index]*(-dw_acc_dev[index] + ddt_dev[index] + ddu_dev[index]*du_acc_dev[index] + ddv_dev[index]*dv_acc_dev[index]);

    //Solve pu
    puu += sigma_puvx_dev[index]*lambda_i*gradu1_dev[index];
    puv += sigma_puvy_dev[index]*lambda_i*gradu2_dev[index];

    //Solve pv
    pvu += sigma_puvx_dev[index]*lambda_i*gradv1_dev[index];
    pvv += sigma_puvy_dev[index]*lambda_i*gradv2_dev[index];

    //Solve pw
    pwu += sigma_pwx_dev[index]*lambda_d*gradw1_dev[index];
    pwv += sigma_pwy_dev[index]*lambda_d*gradw2_dev[index];

    //Constrain pd
    module_p = fabsf(pd);
    if (module_p > 1.f)
    {
        if (pd > 1.f)
            pd_dev[index] = 1.f;
        else
            pd_dev[index] = -1.f;
    }
	else
		pd_dev[index] = pd;

    //Constrain pu
    module_p = rhypotf(puu, puv);	//1.f/sqrtf(puu*puu + puv*puv);
    if (module_p < 1.f)
    {
        puu_dev[index] = puu*module_p;
        puv_dev[index] = puv*module_p;
    }
	else
	{
        puu_dev[index] = puu;
        puv_dev[index] = puv;
	}

    //Constrain pv
    module_p = rhypotf(pvu, pvv);	//1.f/sqrtf(pvu*pvu + pvv*pvv);
    if (module_p < 1.f)
    {
        pvu_dev[index] = pvu*module_p;
        pvv_dev[index] = pvv*module_p;
    }
	else
	{
        pvu_dev[index] = pvu;
        pvv_dev[index] = pvv;
	}

    //Constrain pw
    module_p = rhypotf(pwu, pwv);	//1.f/sqrt(pwu*pwu + pwv*pwv);
    if (module_p < 1.f)
    {
        pwu_dev[index] = pwu*module_p;
        pwv_dev[index] = pwv*module_p;
    }
	else
	{
        pwu_dev[index] = pwu;
        pwv_dev[index] = pwv;
	}

}

__device__ void CSF_cuda::updatePrimalVariables(unsigned int index)
{    
	float du = du_new_dev[index], dv = dv_new_dev[index], dw = dw_new_dev[index];
    const float du_old = du, dv_old = dv, dw_old = dw;
	
	//Compute du, dv and dw
    //Solve du
	du += - tau_u_dev[index]*(mu_uv_dev[index]*ddu_dev[index]*pd_dev[index] - lambda_i*divpu_dev[index]);

    //Solve dv
    dv += - tau_v_dev[index]*(mu_uv_dev[index]*ddv_dev[index]*pd_dev[index] - lambda_i*divpv_dev[index]);

    //Solve dw
    dw += - tau_w_dev[index]*(-mu_uv_dev[index]*pd_dev[index] - lambda_d*divpw_dev[index]);

    //shrink du, dv and dw
    //-----------------------------------------------------------
    const float optflow = dct_dev[index] + dcu_dev[index]*du + dcv_dev[index]*dv;
    const float of_threshold = tau_u_dev[index]*dcu_dev[index]*dcu_dev[index] + tau_v_dev[index]*dcv_dev[index]*dcv_dev[index];
    if (optflow < -of_threshold)
    {
		du += tau_u_dev[index]*dcu_dev[index];
        dv += tau_v_dev[index]*dcv_dev[index];
    }
    else if (optflow > of_threshold)
    {
		du -= tau_u_dev[index]*dcu_dev[index];
        dv -= tau_v_dev[index]*dcv_dev[index];
    }
    else
    {
        const float den = tau_u_dev[index]*dcu_dev[index]*dcu_dev[index] + tau_v_dev[index]*dcv_dev[index]*dcv_dev[index] + 1e-10f;
		du -= tau_u_dev[index]*dcu_dev[index]*optflow/den;
        dv -= tau_v_dev[index]*dcv_dev[index]*optflow/den;
    }

    //Update du, dv
    du_acc_dev[index] = 2.f*du - du_old;
    dv_acc_dev[index] = 2.f*dv - dv_old;
    dw_acc_dev[index] = 2.f*dw - dw_old;

	du_new_dev[index] = du;
	dv_new_dev[index] = dv;
	dw_new_dev[index] = dw;
}

__device__ void CSF_cuda::computeDivergence(unsigned int index)
{
    //Calculate (v,u)
    const unsigned int v = index%rows_i;
    const unsigned int u = index/rows_i;

    //First terms
    if (u == 0)
    {
        divpu_dev[index] = ri_dev[index]*puu_dev[index];
        divpv_dev[index] = ri_dev[index]*pvu_dev[index];
        divpw_dev[index] = ri_dev[index]*pwu_dev[index];
    }
    else if (u == cols_i-1)
    {
        divpu_dev[index] = -ri_dev[index-rows_i]*puu_dev[index-rows_i];
        divpv_dev[index] = -ri_dev[index-rows_i]*pvu_dev[index-rows_i];
        divpw_dev[index] = -ri_dev[index-rows_i]*pwu_dev[index-rows_i];
    }
    else
    {
        divpu_dev[index] = ri_dev[index]*puu_dev[index] - ri_dev[index-rows_i]*puu_dev[index-rows_i];
        divpv_dev[index] = ri_dev[index]*pvu_dev[index] - ri_dev[index-rows_i]*pvu_dev[index-rows_i];
        divpw_dev[index] = ri_dev[index]*pwu_dev[index] - ri_dev[index-rows_i]*pwu_dev[index-rows_i];
    }

    //Second term
    if (v == 0)
    {
        divpu_dev[index] += rj_dev[index]*puv_dev[index];
        divpv_dev[index] += rj_dev[index]*pvv_dev[index];
        divpw_dev[index] += rj_dev[index]*pwv_dev[index];
    }
    else if (v == rows_i-1)
    {
        divpu_dev[index] += -rj_dev[index-1]*puv_dev[index-1];
        divpv_dev[index] += -rj_dev[index-1]*pvv_dev[index-1];
        divpw_dev[index] += -rj_dev[index-1]*pwv_dev[index-1];
    }
    else
    {
        divpu_dev[index] += rj_dev[index]*puv_dev[index] - rj_dev[index-1]*puv_dev[index-1];
        divpv_dev[index] += rj_dev[index]*pvv_dev[index] - rj_dev[index-1]*pvv_dev[index-1];
        divpw_dev[index] += rj_dev[index]*pwv_dev[index] - rj_dev[index-1]*pwv_dev[index-1];
    }
}

__device__ void CSF_cuda::computeGradient(unsigned int index)
{
    //Calculate (v,u)
    const unsigned int v = index%rows_i;
    const unsigned int u = index/rows_i;

    if (u == cols_i-1)
    {
        gradu1_dev[index] = 0.f;
        gradv1_dev[index] = 0.f;
        gradw1_dev[index] = 0.f;
    }
    else
    {
        gradu1_dev[index] = ri_dev[index]*((du_acc_dev[index+rows_i] + du_prev_dev[index+rows_i]) - (du_acc_dev[index] + du_prev_dev[index]));
        gradv1_dev[index] = ri_dev[index]*((dv_acc_dev[index+rows_i] + dv_prev_dev[index+rows_i]) - (dv_acc_dev[index] + dv_prev_dev[index]));
        gradw1_dev[index] = ri_dev[index]*(dw_acc_dev[index+rows_i] - dw_acc_dev[index]);
    }

    if (v == rows_i-1)
    {
        gradu2_dev[index] = 0.f;
        gradv2_dev[index] = 0.f;
        gradw2_dev[index] = 0.f;
    }
    else
    {
        gradu2_dev[index] = rj_dev[index]*((du_acc_dev[index+1] + du_prev_dev[index+1]) - (du_acc_dev[index] + du_prev_dev[index]));
        gradv2_dev[index] = rj_dev[index]*((dv_acc_dev[index+1] + dv_prev_dev[index+1]) - (dv_acc_dev[index] + dv_prev_dev[index]));
        gradw2_dev[index] = rj_dev[index]*(dw_acc_dev[index+1] - dw_acc_dev[index]);
    }
}


//                              Filter
//=============================================================================
__device__ void CSF_cuda::saturateVariables(unsigned int index)
{
    float du = du_new_dev[index], dv = dv_new_dev[index], dw = dw_new_dev[index];
	if (du > 1.f)
        du = 1.f;
    else if (du < -1.f)
        du = -1.f;

    if (dv > 1.f)
        dv = 1.f;
    else if (dv < -1.f)
        dv = -1.f;

    if (depth_old_dev[level_image][index] == 0.f)
        dw = 0.f;

	//Add previous solution to filter all together
	du_new_dev[index] = du + du_prev_dev[index];
	dv_new_dev[index] = dv + dv_prev_dev[index];
	dw_new_dev[index] = dw;
}

__device__ void CSF_cuda::filterSolution(unsigned int index)
{
	const float depth_old = depth_old_dev[level_image][index];
	
	//Calculate (v,u)
    const unsigned int v = index%rows_i;
    const unsigned int u = index/rows_i;

	//								Weighted median filter
	//----------------------------------------------------------------------------------------
	fieldAndPresence up[9], vp[9], wp[9];
    float pres_cum_u[9], pres_cum_v[9], pres_cum_w[9], pres_med;
    int indr, indc, ind_loop;
	unsigned int point_count, v_index;
    const float kd = 5.f;
    const float kddt = 10.f;

	if (depth_old > 0.f)
    {
        point_count = 9;
        v_index = 0;

        for (int k=-1; k<2; k++)
            for (int l=-1; l<2; l++)
            {
                indr = v+k;
                indc = u+l;
				ind_loop = index + l*rows_i + k;
                if ((indr < 0)||(indr >= rows_i)||(indc < 0)||(indc >= cols_i))
                {
                    point_count--;
                    continue;
                }

                //Compute weights
                const float pres = 1.f/(1.f + kd*powf(depth_old - depth_old_dev[level_image][ind_loop],2.f) + kddt*powf(ddt_dev[ind_loop],2.f));
						
				up[v_index].field = du_new_dev[ind_loop]; up[v_index].pres = pres;
                vp[v_index].field = dv_new_dev[ind_loop]; vp[v_index].pres = pres;
                wp[v_index].field = dw_new_dev[ind_loop]; wp[v_index].pres = pres;
                v_index++;
            }

		//Sort vectors (both the solution and the weights)
		bubbleSortDev(up, point_count);
		bubbleSortDev(vp, point_count);
		bubbleSortDev(wp, point_count);

		//Compute cumulative weight
		pres_cum_u[0] = up[0].pres; pres_cum_v[0] = vp[0].pres; pres_cum_w[0] = wp[0].pres;
		for (unsigned int i=1; i<point_count; i++)
		{
			pres_cum_u[i] = pres_cum_u[i-1] + up[i].pres;
			pres_cum_v[i] = pres_cum_v[i-1] + vp[i].pres;
			pres_cum_w[i] = pres_cum_w[i-1] + wp[i].pres;
		}
				
		pres_med = 0.5f*pres_cum_u[point_count-1];

		//Look for the indices comprising pres_med and get the filtered value
		unsigned int cont = 0, ind_l, ind_r;

		//For u
		while (pres_med > pres_cum_u[cont]) {cont++;}
		if (cont == 0)
            du_l_dev[index] = up[0].field;
		else
		{
			ind_r = cont; ind_l = cont-1;
            du_l_dev[index] = ((pres_cum_u[ind_r] - pres_med)*up[ind_l].field + (pres_med - pres_cum_u[ind_l])*up[ind_r].field)/(pres_cum_u[ind_r] - pres_cum_u[ind_l]);
		}

		//For v
		cont = 0;
		while (pres_med > pres_cum_v[cont]) {cont++;}
		if (cont == 0)
            dv_l_dev[index] = vp[0].field;
		else
		{
			ind_r = cont; ind_l = cont-1;
            dv_l_dev[index] = ((pres_cum_v[ind_r] - pres_med)*vp[ind_l].field + (pres_med - pres_cum_v[ind_l])*vp[ind_r].field)/(pres_cum_v[ind_r] - pres_cum_v[ind_l]);
		}

		//For w
		cont = 0;
		while (pres_med > pres_cum_w[cont]) {cont++;}
		if (cont == 0)
            dw_l_dev[index] = wp[0].field;
		else
		{
			ind_r = cont; ind_l = cont-1;
            dw_l_dev[index] = ((pres_cum_w[ind_r] - pres_med)*wp[ind_l].field + (pres_med - pres_cum_w[ind_l])*wp[ind_r].field)/(pres_cum_w[ind_r] - pres_cum_w[ind_l]);
		}
    }
	else
	{
        du_l_dev[index] = du_new_dev[index];
        dv_l_dev[index] = dv_new_dev[index];
        dw_l_dev[index] = dw_new_dev[index];
	}

    pd_l_dev[index] = pd_dev[index];
    puu_l_dev[index] = puu_dev[index];
    puv_l_dev[index] = puv_dev[index];
    pvu_l_dev[index] = pvu_dev[index];
    pvv_l_dev[index] = pvv_dev[index];
    pwu_l_dev[index] = pwu_dev[index];
    pwv_l_dev[index] = pwv_dev[index];
}

__device__ void CSF_cuda::computeMotionField(unsigned int index)
{
	const float inv_f = 2.f*tanf(0.5f*fovh)/float(cols);

    //Fill the matrices dx,dy,dz with the scene flow estimate
    if (depth_old_dev[level_image][index] > 0)
    {
        dx_dev[index] = dw_l_dev[index];
        dy_dev[index] = depth_old_dev[level_image][index]*du_l_dev[index]*inv_f + dw_l_dev[index]*xx_old_dev[level_image][index]/depth_old_dev[level_image][index];
        dz_dev[index] = depth_old_dev[level_image][index]*dv_l_dev[index]*inv_f + dw_l_dev[index]*yy_old_dev[level_image][index]/depth_old_dev[level_image][index];
    }
    else
    {
        dx_dev[index] = 0.f;
        dy_dev[index] = 0.f;
        dz_dev[index] = 0.f;
    }
}


//                              Bridges
//=================================================================================
void GaussianPyramidBridge(CSF_cuda *csf, unsigned int levels, unsigned int cam_mode)
{
    for (unsigned int i=0; i<levels; i++)
    {
        const unsigned int cols_i_aux = 640/(cam_mode*powf(2,i));
        const unsigned int rows_i_aux = 480/(cam_mode*powf(2,i));

        hipMemcpy(&csf->rows_i, &rows_i_aux, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&csf->cols_i, &cols_i_aux, sizeof(float), hipMemcpyHostToDevice);

        ComputePyramidLevelKernel <<<N_blocks, N_threads>>>(csf, i);
    }
}

void AssignZerosBridge(CSF_cuda *csf)
{
    AssignZerosKernel <<<N_blocks, N_threads>>>(csf);
}

void UpsampleBridge(CSF_cuda *csf)
{
    UpsampleCopyKernel <<<N_blocks, N_threads>>>(csf);
    UpsampleFilterKernel <<<N_blocks, N_threads>>>(csf);
}

void ImageGradientsBridge(CSF_cuda *csf)
{
    ComputeImGradients <<<N_blocks, N_threads>>>(csf);
}

void WarpingBridge(CSF_cuda *csf)
{
    PerformWarping <<<N_blocks, N_threads>>> (csf);
}

void RijBridge(CSF_cuda *csf)
{
    RijKernel <<<N_blocks, N_threads>>>(csf);
}

void MuAndStepSizesBridge(CSF_cuda *csf)
{
    MuAndStepSizesKernel <<<N_blocks, N_threads>>>(csf);
}

void DualVariablesBridge(CSF_cuda *csf)
{
    DualIteration <<<N_blocks, N_threads>>>(csf);
}

void PrimalVariablesBridge(CSF_cuda *csf)
{
    PrimalIteration <<<N_blocks, N_threads>>>(csf);
}

void DivergenceBridge(CSF_cuda *csf)
{
    DivergenceComputation <<<N_blocks, N_threads>>>(csf);
}

void GradientBridge(CSF_cuda *csf)
{
    GradientComputation <<<N_blocks, N_threads>>>(csf);
}

void FilterBridge(CSF_cuda *csf)
{
    SaturateSolution <<<N_blocks, N_threads>>>(csf);
    FilterSolution <<<N_blocks, N_threads>>>(csf);
}

void MotionFieldBridge(CSF_cuda *csf)
{
    MotionFieldKernel <<<N_blocks, N_threads>>>(csf);
}

void DebugBridge(CSF_cuda *csf_device)
{
    printf("Executing debug kernel");
    DebugKernel <<<1,1>>>(csf_device);
}

void BridgeBack(CSF_cuda *csf_host, CSF_cuda *csf_device)
{
    hipMemcpy(csf_host, csf_device, sizeof(CSF_cuda), hipMemcpyDeviceToHost);
    hipFree(csf_device);
}


//                                  Kernels
//=============================================================================
__global__ void DebugKernel(CSF_cuda *csf)
{
    //Add here the code you want to use for debugging
	printf("\n dx: ");
    for (unsigned int i = 0; i< (csf->rows_i)*(csf->cols_i); i++)
        printf(" %f", csf->dx_dev[i]);

}

__global__ void ComputePyramidLevelKernel (CSF_cuda *csf, unsigned int level)
{
    // detect pixel
    unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;

    while (index < csf->rows_i*csf->cols_i)
    {
        csf->computePyramidLevel(index, level);
        index += blockDim.x*gridDim.x;
    }
}

__global__ void AssignZerosKernel (CSF_cuda *csf)
{
    // detect pixel
    unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;

    while (index < csf->rows_i*csf->cols_i)
    {
        csf->assignZeros(index);
        index += blockDim.x*gridDim.x;
    }
}

__global__ void UpsampleCopyKernel (CSF_cuda *csf)
{
    // detect pixel
    unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;

    while (index < csf->rows_i*csf->cols_i/4)
    {
        csf->upsampleCopyPrevSolution(index);
        index += blockDim.x*gridDim.x;
    }
}

__global__ void UpsampleFilterKernel (CSF_cuda *csf)
{
    // detect pixel
    unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;

    while (index < csf->rows_i*csf->cols_i)
    {
        csf->upsampleFilterPrevSolution(index);
        index += blockDim.x*gridDim.x;
    }
}

__global__ void ComputeImGradients(CSF_cuda *csf)
{
    // detect pixel
    unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;

    while (index < csf->rows_i*csf->cols_i)
    {
        csf->computeImGradients(index);
        index += blockDim.x*gridDim.x;
    }
}

__global__ void PerformWarping(CSF_cuda *csf)
{
    // detect pixel
    unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;

    while (index < csf->rows_i*csf->cols_i)
    {
        csf->performWarping(index);
        index += blockDim.x*gridDim.x;
    }
}

__global__ void RijKernel(CSF_cuda *csf)
{
    // detect pixel
    unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;

    while (index < csf->rows_i*csf->cols_i)
    {
        csf->computeRij(index);
        index += blockDim.x*gridDim.x;
    }
}

__global__ void MuAndStepSizesKernel(CSF_cuda *csf)
{
    // detect pixel
    unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;

    while (index < csf->rows_i*csf->cols_i)
    {
        csf->computeMu(index);
        csf->computeStepSizes(index);
        index += blockDim.x*gridDim.x;
    }
}

__global__ void DualIteration(CSF_cuda *csf)
{
    // detect pixel
    unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;

    while (index < csf->rows_i*csf->cols_i)
    {
        csf->updateDualVariables(index);
        index += blockDim.x*gridDim.x;
    }
}

__global__ void PrimalIteration(CSF_cuda *csf)
{
    // detect pixel
    unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;

    while (index < csf->rows_i*csf->cols_i)
    {
        csf->updatePrimalVariables(index);
        index += blockDim.x*gridDim.x;
    }
}

__global__ void DivergenceComputation(CSF_cuda *csf)
{
    // detect pixel
    unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;

    while (index < csf->rows_i*csf->cols_i)
    {
        csf->computeDivergence(index);
        index += blockDim.x*gridDim.x;
    }
}


__global__ void GradientComputation(CSF_cuda *csf)
{
    // detect pixel
    unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;

    while (index < csf->rows_i*csf->cols_i)
    {
        csf->computeGradient(index);
        index += blockDim.x*gridDim.x;
    }
}


__global__ void SaturateSolution (CSF_cuda *csf)
{
    // detect pixel
    unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;

    while (index < csf->rows_i*csf->cols_i)
    {
        csf->saturateVariables(index);
        index += blockDim.x*gridDim.x;
    }
}


__global__ void FilterSolution (CSF_cuda *csf)
{
    // detect pixel
    unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;

    while (index < csf->rows_i*csf->cols_i)
    {
        csf->filterSolution(index);
        index += blockDim.x*gridDim.x;
    }
}

__global__ void MotionFieldKernel (CSF_cuda *csf)
{
    // detect pixel
    unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;

    while (index < csf->rows_i*csf->cols_i)
    {
        csf->computeMotionField(index);
        index += blockDim.x*gridDim.x;
    }
}

//Naive implementations of bubbleSort (applied to very small arrays)
__device__ void bubbleSortDev(fieldAndPresence array[], unsigned int num_elem)
{
	bool go_on = true;
	while (go_on)
	{
		go_on = false;
		for (unsigned int i=1; i<num_elem; i++)
		{
			if (array[i-1].field > array[i].field)
			{
				ELEM_SWAP(array[i-1].field,array[i].field);
				ELEM_SWAP(array[i-1].pres,array[i].pres);
				go_on = true;
			}
		}
	}
}
